#include "hip/hip_runtime.h"
//Edit by Piaobo
//data:2023.2.15
#include "ntt.h" 	
using namespace std;
void cpuToGpuMemcpy(uint64_t* h_data, uint64_t* d_data, int size)
{
    hipError_t err = hipMemcpy(d_data, h_data, size, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector from host device! - %s", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

void gpuToCpuMemcpy(uint64_t* d_data, uint64_t* h_data, int size)
{
    hipError_t err = hipMemcpy(h_data, d_data, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector from gpu device! - %s", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    hipFree(d_data);
}


uint64_t* ParallelNTT2D(uint64_t* vec, uint64_t batchSize, uint64_t Len_1D, uint64_t Len_2D, uint64_t* twiddleFactorArray2D_coef, uint64_t p, uint64_t G, uint64_t* twiddleFactorArray_1st, uint64_t* twiddleFactorArray_2nd, bool rev) {       
    
    uint64_t* result; // , * result_cpu;
    //uint64_t w, k_, a;
    //uint64_t factor1, factor2;
    uint64_t n = Len_1D * Len_2D;
    

    uint64_t* proc_twiddleFactorArray_1st = NULL;
    uint64_t* proc_twiddleFactorArray_2nd = NULL;
     uint64_t* proc_twiddleFactorArray2D_coef = NULL;

    result = (uint64_t*)malloc(n * batchSize * sizeof(uint64_t));
    //result_cpu = (uint64_t*)malloc(n * batchSize * sizeof(uint64_t));
    proc_twiddleFactorArray2D_coef = (uint64_t*)malloc(n * batchSize * sizeof(uint64_t));
    uint64_t* result_Temp;

    uint64_t nbits = log2(Len_2D);
    uint64_t reNumber = 0;
    uint64_t preNumber = 0;
   
    if(rev)
    {
        for (uint64_t ir = 0; ir < Len_2D; ir++)
        {
            result_Temp = bit_reverse(vec + ir * Len_1D, Len_1D);
            memcpy(result + ir * Len_1D, result_Temp, Len_1D *sizeof(uint64_t));
            

            bit_reverseOfNumber(&ir, &nbits, &reNumber);
            memcpy(proc_twiddleFactorArray2D_coef + reNumber * Len_1D, twiddleFactorArray2D_coef + ir * Len_1D, Len_1D * sizeof(uint64_t));

        }
        free(result_Temp);      
        proc_twiddleFactorArray_1st = twiddleFactorArray_1st;
        proc_twiddleFactorArray_2nd = twiddleFactorArray_2nd;
    }
    else
    {
        for(uint64_t i = 0; i < n; i++)
        {
            result[i] = vec[i];
            //result_cpu[i] = vec[i];
        }

        uint32_t Stage_1st = log2(Len_1D);
        uint32_t Stage_2nd = log2(Len_2D);
        proc_twiddleFactorArray_1st = (uint64_t*)malloc(Stage_1st * Len_1D/2 * sizeof(uint64_t));
        proc_twiddleFactorArray_2nd = (uint64_t*)malloc(Stage_2nd * Len_2D/2 * sizeof(uint64_t));

        uint32_t wLen_1D = Len_1D / 2;
        for (uint32_t ir = 0; ir < Stage_1st; ir++)
        {
            memcpy(proc_twiddleFactorArray_1st+ (Stage_1st-1- ir) * wLen_1D, twiddleFactorArray_1st+ir* wLen_1D, wLen_1D * sizeof(uint64_t));
        }

        uint32_t wLen_2D = Len_2D / 2;
        for (uint32_t ir = 0; ir < Stage_2nd; ir++)
        {
            memcpy(proc_twiddleFactorArray_2nd + (Stage_2nd - 1 - ir) * wLen_2D, twiddleFactorArray_2nd + ir * wLen_2D, wLen_2D * sizeof(uint64_t));
        }

        for (uint64_t ir = 0; ir < Len_2D; ir++)
        {
            result_Temp = bit_reverse(twiddleFactorArray2D_coef + ir * Len_1D, Len_1D);
            memcpy(proc_twiddleFactorArray2D_coef + ir * Len_1D, result_Temp, Len_1D * sizeof(uint64_t));
         }
        free(result_Temp);
        
    }


    uint64_t sizeOfRes = Len_1D * Len_2D * sizeof(uint64_t);

    uint64_t* cudatwiddleFactorArray_1st = NULL;
    uint64_t* cudatwiddleFactorArray_2nd = NULL;
    uint64_t* cudatwiddleFactorArray2D_coef = NULL;
    uint64_t wcoffLen1 = log2(Len_1D) * Len_1D / 2 * sizeof(uint64_t);
    uint64_t wcoffLen2 = log2(Len_2D) * Len_2D / 2 * sizeof(uint64_t);
    hipMalloc(&cudatwiddleFactorArray_1st, wcoffLen1);
    hipMalloc(&cudatwiddleFactorArray_2nd, wcoffLen2);
    hipMalloc(&cudatwiddleFactorArray2D_coef, sizeOfRes);
    cpuToGpuMemcpy(proc_twiddleFactorArray_1st, cudatwiddleFactorArray_1st, wcoffLen1);
    cpuToGpuMemcpy(proc_twiddleFactorArray_2nd, cudatwiddleFactorArray_2nd, wcoffLen2);
    cpuToGpuMemcpy(proc_twiddleFactorArray2D_coef, cudatwiddleFactorArray2D_coef, sizeOfRes);

  hipEvent_t start, stop;
  float elapsedTime;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);
  
  //printf("GPU 1D Implementation \n");
    //GPU implementation 1D

  uint64_t* outMatrix_1st;
  uint64_t* outMatrix_mid;
  uint64_t* outMatrix_2nd;
  hipMalloc(&outMatrix_1st, sizeOfRes);
  hipMalloc(&outMatrix_mid, sizeOfRes);
  hipMalloc(&outMatrix_2nd, sizeOfRes);
  cpuToGpuMemcpy(result, outMatrix_1st, sizeOfRes);
  //cpuToGpuMemcpy(result, outMatrix_mid, sizeOfRes);
  //cpuToGpuMemcpy(result, outMatrix_2nd, sizeOfRes);

  hipStream_t streams[NUM_STREAMS];
  //float* data[NUM_STREAMS];

  for (uint64_t ir = 0; ir < NUM_STREAMS; ir++)
  {
      hipStreamCreate(&streams[ir]);
  }

    for (uint64_t ir = 0; ir < Len_2D; ir+= NUM_STREAMS)
    {
        //cuda_ntt_parallelNew(outMatrix_1st + Len_1D * ir, outMatrix_2nd + Len_1D * ir, batchSize, Len_1D, p, G, log2(Len_1D), cudatwiddleFactorArray_1st);
        for (uint64_t irK = 0; irK < NUM_STREAMS; irK++)
        {
            cuda_ntt_parallelNew(outMatrix_1st + Len_1D * (ir + irK), outMatrix_2nd + Len_1D * (ir + irK), batchSize, Len_1D, p, G, log2(Len_1D), Len_1D>>1, cudatwiddleFactorArray_1st, &streams[irK]);
        }
        //cuda_ntt_parallelNew(outMatrix_1st + Len_1D * ir, outMatrix_2nd + Len_1D * ir, batchSize, Len_1D, p, G, log2(Len_1D), cudatwiddleFactorArray_1st, &streams[0]);
        //cuda_ntt_parallelNew(outMatrix_1st + Len_1D * (ir + 1), outMatrix_2nd + Len_1D * (ir + 1), batchSize, Len_1D, p, G, log2(Len_1D), cudatwiddleFactorArray_1st, &streams[1]);
        //cuda_ntt_parallelNew(outMatrix_1st + Len_1D * (ir + 2), outMatrix_2nd + Len_1D * (ir + 2), batchSize, Len_1D, p, G, log2(Len_1D), cudatwiddleFactorArray_1st, &streams[2]);
        //cuda_ntt_parallelNew(outMatrix_1st + Len_1D * (ir + 3), outMatrix_2nd + Len_1D * (ir + 3), batchSize, Len_1D, p, G, log2(Len_1D), cudatwiddleFactorArray_1st, &streams[3]);
        //cuda_DataTranspose << <1, 1 >> > (0, 0, 0, 0);
        
    }
    hipDeviceSynchronize();

    //hipFree(DataIn);
    //hipFree(DataOut);

    //printf("GPU 1D Implementation end \n");
    
    //printf("Multiply by Twiddle Factors \n");
    //void* kernelArgs[] = {
    //(void*)&outMatrix_mid, (void*)&outMatrix_2nd, (void*)&cudatwiddleFactorArray2D_coef,(void*)&n, (void*)&p
    //};
    //hipLaunchCooperativeKernel((void*)cuda_NTTStep2, MAX_GRID, THREDS_PER_BLOCK, kernelArgs);//����ͬ��
    //hipDeviceSynchronize();
    // 
    // 
    uint64_t tpb = 256;
    uint64_t bpg = (batchSize * n - 1 + tpb) / tpb; // Blocks per grid
    dim3 dimBlock2(tpb, 1, 1);
    //int grid_2D = pow(2, log2(bpg) / 2); 
    //dim3 dimGrid2(grid_2D, grid_2D, 1);
    //dim3 dimGrid2(256, (bpg - 1 + 256) / 256, 1);
    dim3 dimGrid2(n / tpb, 1, 1);
    //if (bpg<=256)
    //{
    //    dim3 dimGrid2(bpg, 1, 1);
    //}
    //else
    //{
    //    dim3 dimGrid2(256, (bpg - 1 + 256) / 256, 1);
    //}
    
    cuda_NTTStep2_w << <dimGrid2, dimBlock2 >> > (outMatrix_2nd, outMatrix_mid, cudatwiddleFactorArray2D_coef, n, p);
    hipDeviceSynchronize();
    //printf("Multiply by Twiddle Factors end \n");


   // printf("Data Transpose \n");

    cuda_DataTranspose_W << <dimGrid2, dimBlock2 >> > (outMatrix_mid, outMatrix_2nd, Len_1D, Len_2D);
    hipDeviceSynchronize();

    //void* kernelArgs2[] = {
    //(void*)&outMatrix_mid, (void*)&outMatrix_2nd, (void*)&Len_1D, (void*)&Len_2D
    //};
    //hipLaunchCooperativeKernel((void*)cuda_DataTranspose, MAX_GRID, THREDS_PER_BLOCK, kernelArgs2);//����ͬ��
    //hipDeviceSynchronize();    

    //cuda_DataTranspose<<<MAX_GRID , THREDS_PER_BLOCK >>>(outMatrix_mid, outMatrix_2nd, Len_1D, Len_2D);

    //printf("Data Transpose end \n");

    //GPU implementation 2D

    //uint64_t* resultcpy;
    //resultcpy = (uint64_t*)malloc(Len_2D * sizeof(uint64_t));
    //memcpy(resultcpy, result + Len_2D * 0, Len_2D * sizeof(uint64_t));
    //uint64_t* outVec4 = ParallelNTT(resultcpy, batchSize, Len_2D, p, G, proc_twiddleFactorArray_2nd, 0);
    //printf("GPU 2D Implementation\n");
    for (uint64_t ir = 0; ir < Len_1D; ir+= NUM_STREAMS)
    {
        //cuda_ntt_parallelNew(outMatrix_2nd + Len_2D * ir, outMatrix_mid + Len_2D * ir, batchSize, Len_2D, p, G, log2(Len_2D), cudatwiddleFactorArray_2nd);
        for (uint64_t irK = 0; irK < NUM_STREAMS; irK++)
        {
            cuda_ntt_parallelNew(outMatrix_2nd + Len_2D * (ir + irK), outMatrix_mid + Len_2D * (ir + irK), batchSize, Len_2D, p, G, log2(Len_2D), Len_2D>>1,cudatwiddleFactorArray_2nd, &streams[irK]);
        }
        //cuda_ntt_parallelNew(outMatrix_2nd + Len_2D * ir, outMatrix_mid + Len_2D * ir, batchSize, Len_2D, p, G, log2(Len_2D), cudatwiddleFactorArray_2nd, &streams[0]);
        //cuda_ntt_parallelNew(outMatrix_2nd + Len_2D * (ir + 1), outMatrix_mid + Len_2D * (ir + 1), batchSize, Len_2D, p, G, log2(Len_2D), cudatwiddleFactorArray_2nd, &streams[1]);
        //cuda_ntt_parallelNew(outMatrix_2nd + Len_2D * (ir + 2), outMatrix_mid + Len_2D * (ir + 2), batchSize, Len_2D, p, G, log2(Len_2D), cudatwiddleFactorArray_2nd, &streams[2]);
        //cuda_ntt_parallelNew(outMatrix_2nd + Len_2D * (ir + 3), outMatrix_mid + Len_2D * (ir + 3), batchSize, Len_2D, p, G, log2(Len_2D), cudatwiddleFactorArray_2nd, &streams[3]);
        //cuda_DataTranspose << <1, 1 >> > (0, 0, 0, 0);
        
    }
    hipDeviceSynchronize();

    cuda_DataTranspose_W << <dimGrid2, dimBlock2 >> > (outMatrix_mid, outMatrix_2nd, Len_1D, Len_2D);
    hipDeviceSynchronize();

    //void* kernelArgs3[] = {
    //(void*)&outMatrix_mid, (void*)&outMatrix_2nd, (void*)&Len_2D, (void*)&Len_1D
    //};
    //hipLaunchCooperativeKernel((void*)cuda_DataTranspose, MAX_GRID, THREDS_PER_BLOCK, kernelArgs3);//����ͬ��
    //hipDeviceSynchronize();

    hipMemcpy(result, outMatrix_2nd, sizeOfRes, hipMemcpyDeviceToHost);
    //printf("GPU 2D Implementation end \n");

    //gpuToCpuMemcpy(outMatrix_mid, result, sizeOfRes);  // debug

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    //printf("GPU Time cost: %3.1f ms", elapsedTime);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipFree(&outMatrix_1st);
    hipFree(&outMatrix_2nd);
    hipFree(&cudatwiddleFactorArray_1st);
    hipFree(&cudatwiddleFactorArray_2nd);
    hipFree(&cudatwiddleFactorArray2D_coef);
    hipFree(&outMatrix_mid);

    //Comparison
    //if (CPURUAN) 
    //{
    //    bool compCPUGPUResult = compVec(result, result_cpu, batchSize * n, false);
    //    std::cout << "\nComparing output of cpu and gpu :" << compCPUGPUResult;
    //}

    return result;

}


// ��䣿
__global__ void cuda_DataTranspose(uint64_t* Res, uint64_t* ResOut, uint64_t Len_1D, uint64_t Len_2D)
{
    uint64_t DataIndex = 0;
    
    for (uint32_t ir = 0; ir < Len_1D; ir++)
    {
        for (uint32_t ir2 = 0; ir2 < Len_2D; ir2++)
        {
            ResOut[DataIndex] = Res[ir + Len_1D * ir2];

            //DataTest = outMatrix_1st[DataIndex];
            //outMatrix_2nd[DataIndex] = outMatrix_mid[DataIndex];
            DataIndex++;
        }
    }
}

__global__ void cuda_DataTranspose_W(uint64_t* Res, uint64_t* ResOut, uint64_t Len_1D, uint64_t Len_2D)
{
    uint64_t row, col;
    uint64_t global_idx = blockDim.x * blockIdx.x + blockDim.y * blockIdx.y + threadIdx.x;//
    if (global_idx< Len_1D* Len_2D)
    {
        col = global_idx % Len_2D;
        row = uint64_t(global_idx / Len_2D);
        ResOut[col * Len_1D + row] = Res[global_idx];
    }
}

__global__ void cuda_NTTStep2(uint64_t* Res, uint64_t* ResOut, uint64_t* coef, uint64_t Len, uint64_t p)
{
    uint64_t DataIndex = 0;
    uint128_t Temp = 0;
    uint64_t TempMod = 0;

    for (uint64_t ir2 = 0; ir2 < Len; ir2++)
    {
        mul64(Res[ir2], coef[ir2], Temp);
        TempMod = (Temp % p).low;
        ResOut[ir2] = TempMod;
    }

}

//2023.6.1 �޸���ת�������
__global__ void cuda_NTTStep2_w(uint64_t* Res, uint64_t* ResOut, uint64_t* coef, uint64_t Len, uint64_t p)
{
    uint64_t global_idx = blockDim.x * blockIdx.x + blockDim.y * blockIdx.y + threadIdx.x;//ȫ���߳����
    /*uint128_t Temp = 0;
    uint64_t TempMod = 0;*/
    if (global_idx < Len)
    {
        /*mul64(Res[global_idx], coef[global_idx], Temp);
        TempMod = (Temp % p).low;
        ResOut[global_idx] = TempMod;*/

        mul64modAdd(Res[global_idx], coef[global_idx], 0, p, ResOut[global_idx]);
    }

}


void cuda_ntt_parallelNew(uint64_t* res, uint64_t* resOut, uint64_t batchSize, uint64_t n, uint64_t p, uint64_t r, uint64_t log2n, uint64_t maxTwiddleCols, uint64_t* twiddleFactorArray, hipStream_t* userStream)
{
    uint64_t* cuda_result, * cuda_output;
    //uint64_t sizeOfRes = batchSize * n * sizeof(uint64_t);
    uint64_t* preComputeTFarray;

    preComputeTFarray = twiddleFactorArray;
    cuda_result = res;
    cuda_output = resOut;
    //preComputeTFarray = twiddleFactorArray;


    // Number of threads my_kernel will be launched with
    uint64_t tpb = THREDS_PER_BLOCK;
    uint64_t bpg = (batchSize * n - 1 + tpb) / tpb; // Blocks per grid

    if (bpg > MAX_GRID)
        bpg = MAX_GRID;

    //    cout<<"bpg: "<<bpg<<endl;

    dim3 dimGrid(bpg, 1, 1);
    dim3 dimBlock(tpb, 1, 1);
    //��ͬ�ĺ˺���
    //ntt parallel compute
    if (KENEL_FUNCTION == 0)
    {
        void* kernelArgs[] = {
        (void*)&cuda_result, (void*)&cuda_output, (void*)&batchSize, (void*)&n,(void*)&p, (void*)&r, (void*)&log2n,
        (void*)&preComputeTFarray
        };
        hipLaunchCooperativeKernel((void*)cuda_ntt_parallel_kernel, dimGrid, dimBlock, kernelArgs, 0, *userStream);//����ͬ��
    }
    if (KENEL_FUNCTION ==1)
    {
        void* kernelArgs[] = {
        (void*)&cuda_result, (void*)&cuda_output, (void*)&batchSize, (void*)&n,(void*)&p, (void*)&r, (void*)&log2n, (void*)&maxTwiddleCols,
        (void*)&preComputeTFarray
        };
        hipLaunchCooperativeKernel((void*)cuda_ntt_parallel_kernelNew, dimGrid, dimBlock, kernelArgs, 0, *userStream);//����ͬ��
    }
   //
    //hipDeviceSynchronize();

}


void cuda_ntt_parallelNew(uint64_t* res, uint64_t* resOut, uint64_t batchSize, uint64_t n, uint64_t p, uint64_t r, uint64_t log2n, uint64_t* twiddleFactorArray)
{
    uint64_t* cuda_result, * cuda_output;
    //uint64_t sizeOfRes = batchSize * n * sizeof(uint64_t);
    uint64_t* preComputeTFarray;

    preComputeTFarray = twiddleFactorArray;
    cuda_result = res;
    cuda_output = resOut;
    //preComputeTFarray = twiddleFactorArray;

    /*hipMalloc(&cuda_result, sizeOfRes);
    hipMalloc(&cuda_output, sizeOfRes);
    hipMalloc(&preComputeTFarray, log2(n) * (n / 2) * sizeof(uint64_t));
    cpuToGpuMemcpy(res, cuda_result, sizeOfRes);
    cpuToGpuMemcpy(twiddleFactorArray, preComputeTFarray, log2(n) * (n / 2) * sizeof(uint64_t));*/

    // Number of threads my_kernel will be launched with
    int tpb = THREDS_PER_BLOCK;
    int bpg = (batchSize * n - 1 + tpb) / tpb; // Blocks per grid

    if (bpg > MAX_GRID)
        bpg = MAX_GRID;

    //    cout<<"bpg: "<<bpg<<endl;

    dim3 dimGrid(bpg, 1, 1);
    dim3 dimBlock(tpb, 1, 1);
    void* kernelArgs[] = {
    (void*)&cuda_result, (void*)&cuda_output, (void*)&batchSize, (void*)&n,(void*)&p, (void*)&r, (void*)&log2n,
    (void*)&preComputeTFarray
    };

   /* hipEvent_t start, stop;
    float elapsedTime;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);*/

    //ntt parallel compute
    hipLaunchCooperativeKernel((void*)cuda_ntt_parallel_kernel, dimGrid, dimBlock, kernelArgs);//����ͬ��
    //hipDeviceSynchronize();

    //hipEventRecord(stop, 0);
    //hipEventSynchronize(stop);
    //hipEventElapsedTime(&elapsedTime, start, stop);
    //printf("GPU Time cost: %3.1f ms", elapsedTime);
    //hipEventDestroy(start);
    //hipEventDestroy(stop);

    hipError_t err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Issues in running the kernel. (%s)", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    //gpuToCpuMemcpy(cuda_output, res, sizeOfRes);
    //hipFree(cuda_result);
    //hipFree(preComputeTFarray);
}



uint64_t* ParallelNTT(uint64_t* vec, uint64_t batchSize, uint64_t n, uint64_t p, uint64_t G, uint64_t* twiddleFactorArray, bool rev) {

    uint64_t* result, * result_cpu;
    uint64_t w, k_, a;
    uint64_t factor1, factor2;
    result = (uint64_t*)malloc(n * batchSize * sizeof(uint64_t));
    result_cpu = (uint64_t*)malloc(n * batchSize * sizeof(uint64_t));

    if (rev) {
        result = bit_reverse(vec, n);
        result_cpu = bit_reverse(vec, n);
    }
    else {
        for (uint64_t i = 0; i < n; i++) {
            result[i] = vec[i];
            result_cpu[i] = vec[i];
        }
    }

    //result = bit_reverse(vec, n);
    //result_cpu = bit_reverse(vec, n);

    //CPU implementation

    if (CPURUAN)
    {
        for (int y = 0; y < batchSize; y++)
        {
            for (uint64_t mid = 1, BitShiftNum = 1; mid < n; mid = mid << 1, BitShiftNum++)
            {
                k_ = (p - 1) >> BitShiftNum;
                a = modExp(G, k_, p);
                for (uint64_t j = 0; j < n; j += (mid << 1))
                {
                    w = 1;
                    for (uint64_t k = 0; k < mid; k++)
                    {
                        factor1 = result_cpu[y * n + j + k];
                        uint128_t tmp;
                        mul64(w, result_cpu[y * n + j + k + mid], tmp);
                        factor2 = (tmp % p).low;
                        result_cpu[y * n + j + k] = ((uint128_t(factor1) + factor2) % p).low;
                        result_cpu[y * n + j + k + mid] = ((uint128_t(factor1) + p - factor2) % p).low;
                        mul64(w, a, tmp);
                        w = (tmp % p).low;
                    }
                }
            }
        }
        //std::cout << std::endl << '\n';
        //printVec(result_cpu, n);
    }


    //GPU implementation
    cuda_ntt_parallel(result, batchSize, n, p, G, log2(n), twiddleFactorArray);

    //uint128_t tmp = 0;
    //for (uint32_t ir = 0; ir < n; ir++)
    //{
    //    std::cout << result[ir] << std::endl;
    //    tmp = tmp + uint128_t(result[ir]);
    //}
    //uint64_t tmpOut = (tmp % p).low;
    //std::cout << "Cmp Sum: " << "\n" << tmpOut << std::endl;

    //Comparison
    if (CPURUAN)
    {
        bool compCPUGPUResult = compVec(result, result_cpu, batchSize * n, false);
        std::cout << "\nComparing output of cpu and gpu :" << compCPUGPUResult << std::endl;
    }
    return result;

}


void cuda_ntt_parallel(uint64_t* res, uint64_t batchSize, uint64_t n, uint64_t p, uint64_t r, uint64_t log2n, uint64_t* twiddleFactorArray)
{
    uint64_t* cuda_result, * cuda_output;
    uint64_t sizeOfRes = batchSize * n * sizeof(uint64_t);
    uint64_t* preComputeTFarray;
    hipMalloc(&cuda_result, sizeOfRes);
    hipMalloc(&cuda_output, sizeOfRes);
    hipMalloc(&preComputeTFarray, log2(n) * (n / 2) * sizeof(uint64_t));
    cpuToGpuMemcpy(res, cuda_result, sizeOfRes);
    cpuToGpuMemcpy(twiddleFactorArray, preComputeTFarray, log2(n) * (n / 2) * sizeof(uint64_t));

    // Number of threads my_kernel will be launched with
    int tpb = THREDS_PER_BLOCK; 
    int bpg = (batchSize * n - 1 + tpb) / tpb; // Blocks per grid

    if (bpg > MAX_GRID)
        bpg = MAX_GRID;

    //    cout<<"bpg: "<<bpg<<endl;

    dim3 dimGrid(bpg, 1, 1);
    dim3 dimBlock(tpb, 1, 1);
    void* kernelArgs[] = {
    (void*)&cuda_result, (void*)&cuda_output, (void*)&batchSize, (void*)&n,(void*)&p, (void*)&r, (void*)&log2n,
    (void*)&preComputeTFarray
    };

    hipEvent_t start, stop;
    float elapsedTime;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);

    //ntt parallel compute
    hipLaunchCooperativeKernel((void*)cuda_ntt_parallel_kernel, dimGrid, dimBlock, kernelArgs);//����ͬ��
    hipDeviceSynchronize();

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("GPU Time cost: %3.1f ms \n", elapsedTime);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipError_t err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Issues in running the kernel. (%s) \n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    gpuToCpuMemcpy(cuda_output, res, sizeOfRes);
    hipFree(cuda_result);
    hipFree(preComputeTFarray);
}

__global__ void cuda_ntt_parallel_kernel(uint64_t* result, uint64_t* output, uint64_t batchSize, uint64_t n, uint64_t p, uint64_t r, uint64_t log2n, uint64_t* twiddleFactorArray)
{
    uint64_t mini_batch_size = blockDim.x * gridDim.x / n;//
    uint64_t num_mini_batches = (batchSize + mini_batch_size - 1) / mini_batch_size;//
    uint64_t mini_batch_offset = mini_batch_size * n;//

    uint64_t global_idx = blockDim.x * blockIdx.x + threadIdx.x;//ȫ���߳����
    uint64_t vec_idx = (blockDim.x * blockIdx.x + threadIdx.x) % n;//�����̺߳�

    uint64_t k, w, k_, a;
    uint64_t factor1, factor2;
    uint64_t m = 1;
    uint128_t tmp;

    cooperative_groups::grid_group grid = cooperative_groups::this_grid();
    uint64_t maxTwiddleCols = n >> 1;

    for (uint64_t mid = 1, BitShiftNum = 1; mid < n; mid = mid << 1, BitShiftNum++)
    {
        k_ = (p - 1) >> BitShiftNum;
        //a = modExp(r, k_, p);
        if (vec_idx < n)
        {
            w = 1;
            k = vec_idx & ((mid << 1) - 1);//  �����ת
            if (k < mid)
            {
                for (int l = 0; l < num_mini_batches; l++)
                {
                    factor1 = result[global_idx + mini_batch_offset * l];

                    uint128_t tmp;
                    mul64(twiddleFactorArray[(BitShiftNum - 1) * maxTwiddleCols + k], result[global_idx + mini_batch_offset * l + mid], tmp);
                    factor2 = (tmp % p).low;
                    output[global_idx + mini_batch_offset * l] = ((uint128_t(factor1) + factor2) % p).low;
                }
            }
            else
            {
                for (int l = 0; l < num_mini_batches; l++)
                {
                    factor1 = result[global_idx + mini_batch_offset * l - mid];
                    uint128_t tmp;
                    mul64(twiddleFactorArray[(BitShiftNum - 1) * maxTwiddleCols + k - mid], result[global_idx + mini_batch_offset * l], tmp);
                    factor2 = (tmp % p).low;
                    output[global_idx + mini_batch_offset * l] = ((uint128_t(factor1) + p - factor2) % p).low;
                }
            }
        }
        grid.sync();//����ͬ��
        if (vec_idx < n)
            for (int l = 0; l < num_mini_batches; l++)
                result[global_idx + mini_batch_offset * l] = output[global_idx + mini_batch_offset * l];
        grid.sync();
    }


}


__global__ void cuda_ntt_parallel_kernelNew(uint64_t* result, uint64_t* output, uint64_t batchSize, uint64_t n, uint64_t p, uint64_t r, uint64_t log2n, uint64_t maxTwiddleCols, uint64_t* twiddleFactorArray)
{

    uint64_t global_idx = blockDim.x * blockIdx.x + threadIdx.x;//ȫ���߳����
    //uint64_t vec_idx = (blockDim.x * blockIdx.x + threadIdx.x) % n;//�����̺߳�
    uint64_t vec_idx = (blockDim.x * blockIdx.x + threadIdx.x);//�����̺߳�
    //uint64_t k, w, k_, a;
    cooperative_groups::grid_group grid = cooperative_groups::this_grid();


    for (uint64_t mid = 1, BitShiftNum = 1; mid < n; mid = mid << 1, BitShiftNum++)
    {
        //k_ = (p - 1) >> BitShiftNum;
        //a = modExp(r, k_, p);
        if (vec_idx < n)
        {
            uint64_t k = vec_idx & ((mid << 1) - 1);//  �����ת
            if (k < mid)
            {
                //for (int l = 0; l < num_mini_batches; l++)
                //{
                //    /*factor1 = result[global_idx + mini_batch_offset * l];
                //    uint128_t tmp;
                //    mul64(twiddleFactorArray[(BitShiftNum - 1) * maxTwiddleCols + k], result[global_idx + mini_batch_offset * l + mid], tmp);
                //    factor2 = (tmp % p).low;
                //    output[global_idx + mini_batch_offset * l] = ((uint128_t(factor1) + factor2) % p).low;*/

                //    mul64modAdd(twiddleFactorArray[(BitShiftNum - 1) * maxTwiddleCols + k], result[global_idx + mini_batch_offset * l + mid], result[global_idx + mini_batch_offset * l], p, output[global_idx + mini_batch_offset * l]);
                //}
                mul64modAdd(twiddleFactorArray[(BitShiftNum - 1) * maxTwiddleCols + k], result[global_idx + mid], result[global_idx], p, output[global_idx]);
            }
            else
            {
                //for (int l = 0; l < num_mini_batches; l++) 
                //{
                //  /*  factor1 = result[global_idx + mini_batch_offset * l - mid];
                //    uint128_t tmp;
                //    mul64(twiddleFactorArray[(BitShiftNum - 1) * maxTwiddleCols + k - mid], result[global_idx + mini_batch_offset * l], tmp);
                //    factor2 = (tmp % p).low;
                //    output[global_idx + mini_batch_offset * l] = ((uint128_t(factor1) + p - factor2) % p).low;*/

                //    mul64modSub(twiddleFactorArray[(BitShiftNum - 1) * maxTwiddleCols + k - mid], result[global_idx + mini_batch_offset * l], result[global_idx + mini_batch_offset * l - mid], p, output[global_idx + mini_batch_offset * l]);

                //}
                mul64modSub(twiddleFactorArray[(BitShiftNum - 1) * maxTwiddleCols + k - mid], result[global_idx], result[global_idx - mid], p, output[global_idx]);
            }

            //grid.sync();//����ͬ��? �Ƿ���Ҫ��
            //result[global_idx] = output[global_idx];
        }
        grid.sync();//����ͬ��
        if (vec_idx < n)
            result[global_idx] = output[global_idx];

        grid.sync();

    }
}

uint64_t* ParallelINTT2D(uint64_t* vec, uint64_t batchSize, uint64_t Len_1D, uint64_t Len_2D, uint64_t* twiddleFactorArray2DInv_coef, uint64_t p, uint64_t G, uint64_t* normCoef, uint64_t* twiddleFactorArrayInv_1st, uint64_t* twiddleFactorArrayInv_2nd, bool rev)
{
    uint64_t n = Len_1D * Len_2D;
    uint64_t sizeOfRes = Len_1D * Len_2D * sizeof(uint64_t);

    //uint64_t* factornormNew = (uint64_t*)calloc(n, sizeof(uint64_t));

    //uint64_t* dataMatrix = DataReform(vec, Len_1D, Len_2D);
    uint64_t* outVec = ParallelNTT2D(vec, batchSize, Len_1D, Len_2D, twiddleFactorArray2DInv_coef, p, G, twiddleFactorArrayInv_1st, twiddleFactorArrayInv_2nd, rev);

    //uint64_t TestTemp;
    //mul64modAdd(outVec[3], normCoef[0], 0, p, TestTemp);

    //std::cout << '\n' << TestTemp << '\n' << std::endl;    

    uint64_t* cuda_DataIn_norm = NULL;
    uint64_t* cuda_DataOut_norm = NULL;
    uint64_t* cuda_factor_norm = NULL;
    hipMalloc(&cuda_DataIn_norm, sizeOfRes);
    hipMalloc(&cuda_DataOut_norm, sizeOfRes);
    hipMalloc(&cuda_factor_norm, sizeOfRes);

    cpuToGpuMemcpy(outVec, cuda_DataIn_norm, sizeOfRes);
    cpuToGpuMemcpy(normCoef, cuda_factor_norm, sizeOfRes);

    //std::cout << outVec[3] << '\n' << std::endl;
    
    uint64_t tpb = 64;
    //uint64_t bpg = (batchSize * n - 1 + tpb) / tpb; // Blocks per grid
    dim3 dimBlock2(tpb, 1, 1);
    dim3 dimGrid2(n / tpb, 1, 1);

    cuda_NTTnorm << < dimGrid2, dimBlock2 >> >(cuda_DataIn_norm, cuda_DataOut_norm, cuda_factor_norm, n, p);
    hipDeviceSynchronize();

    hipError_t err = hipMemcpy(outVec, cuda_DataOut_norm, sizeOfRes, hipMemcpyDeviceToHost);

    //if (err != hipSuccess)
    //{
    //    fprintf(stderr, "Failed to copy vector from gpu device! - %s", hipGetErrorString(err));
    //    //exit(EXIT_FAILURE);
    //}

    //std::cout << '\n'<< outVec[3] << '\n' << std::endl;

    hipFree(cuda_DataIn_norm);
    hipFree(cuda_DataOut_norm);
    hipFree(cuda_factor_norm);

    return outVec;

}

__global__ void cuda_NTTnorm(uint64_t* Res, uint64_t* ResOut, uint64_t* coef, uint64_t Len, uint64_t p)
{
    //uint64_t global_idx = blockDim.x * blockIdx.x + blockDim.y * blockIdx.y + threadIdx.x;//ȫ���߳����

    uint32_t blockId = blockIdx.x + blockIdx.y * gridDim.x;
    uint32_t threadId = blockId * (blockDim.x * blockDim.y)+ (threadIdx.y * blockDim.x) + threadIdx.x;

    uint128_t Temp = 0;
    uint64_t TempMod = 0;
    if (threadId < Len)
    {
        const uint64_t coefLocal = *coef;

        //mul64(Res[threadId], coef[threadId], Temp);
        //TempMod = (Temp % p).low;
        //ResOut[threadId] = TempMod;

        mul64modAdd(Res[threadId], coefLocal, 0, p, ResOut[threadId]);
        //Res[threadId] = ResOut[threadId];

        //ResOut[threadId] = 0;

    }
}


uint64_t* ParallelINTT(uint64_t* vec, uint64_t batchSize, uint64_t n, uint64_t p, uint64_t G, uint64_t* twiddleFactorArray, bool rev) {

    uint64_t* result, * result_cpu;
    uint64_t w, k_, a;
    uint64_t factor1, factor2;
    result = (uint64_t*)malloc(n * batchSize * sizeof(uint64_t));
    result_cpu = (uint64_t*)malloc(n * batchSize * sizeof(uint64_t));

    if (rev) {
        result = bit_reverse(vec, n);
        result_cpu = bit_reverse(vec, n);
    }
    else {
        for (uint64_t i = 0; i < n; i++) {
            result[i] = vec[i];
            result_cpu[i] = vec[i];
        }
    }

    //GPU implementation
    cuda_intt_parallel(result, batchSize, n, p, G, log2(n), twiddleFactorArray);


    return result;

}


void cuda_intt_parallel(uint64_t* res, uint64_t batchSize, uint64_t n, uint64_t p, uint64_t r, uint64_t log2n, uint64_t* twiddleFactorArray)
{
    uint64_t* cuda_result, * cuda_output;
    uint64_t sizeOfRes = batchSize * n * sizeof(uint64_t);
    uint64_t* preComputeTFarray;
    hipMalloc(&cuda_result, sizeOfRes);
    hipMalloc(&cuda_output, sizeOfRes);
    hipMalloc(&preComputeTFarray, log2(n) * (n / 2) * sizeof(uint64_t));
    cpuToGpuMemcpy(res, cuda_result, sizeOfRes);
    cpuToGpuMemcpy(twiddleFactorArray, preComputeTFarray, log2(n) * (n / 2) * sizeof(uint64_t));

    // Number of threads my_kernel will be launched with
    int tpb = THREDS_PER_BLOCK;
    int bpg = (batchSize * n - 1 + tpb) / tpb; // Blocks per grid

    if (bpg > MAX_GRID)
        bpg = MAX_GRID;

    //    cout<<"bpg: "<<bpg<<endl;

    dim3 dimGrid(bpg, 1, 1);
    dim3 dimBlock(tpb, 1, 1);
    void* kernelArgs[] = {
    (void*)&cuda_result, (void*)&cuda_output, (void*)&batchSize, (void*)&n,(void*)&p, (void*)&r, (void*)&log2n,
    (void*)&preComputeTFarray
    };

    hipEvent_t start, stop;
    float elapsedTime;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);

    //ntt parallel compute
    hipLaunchCooperativeKernel((void*)cuda_intt_parallel_kernel, dimGrid, dimBlock, kernelArgs);
    hipDeviceSynchronize();

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("GPU Time cost: %3.1f ms", elapsedTime);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipError_t err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Issues in running the kernel. (%s)", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    gpuToCpuMemcpy(cuda_output, res, sizeOfRes);
    hipFree(cuda_result);
    hipFree(preComputeTFarray);
}

__global__ void cuda_intt_parallel_kernel(uint64_t* result, uint64_t* output, uint64_t batchSize, uint64_t n, uint64_t p, uint64_t r, uint64_t log2n, uint64_t* twiddleFactorArray)
{
    uint64_t mini_batch_size = blockDim.x * gridDim.x / n;//��С�ļ��㵥Ԫ������ÿ��һ������=1��
    uint64_t num_mini_batches = (batchSize + mini_batch_size - 1) / mini_batch_size;//���������� һ���������
    uint64_t mini_batch_offset = mini_batch_size * n;//��Ԫƫ��

    uint64_t global_idx = blockDim.x * blockIdx.x + threadIdx.x;//ȫ���߳����
    uint64_t vec_idx = (blockDim.x * blockIdx.x + threadIdx.x) % n;//�����̺߳�

    uint64_t k, w, k_, a;
    uint64_t factor1, factor2;
    uint64_t m = 1;
    uint128_t tmp;

    cooperative_groups::grid_group grid = cooperative_groups::this_grid();
    uint64_t maxTwiddleCols = n >> 1;

    for (uint64_t mid = 1, BitShiftNum = 1; mid < n; mid = mid << 1, BitShiftNum++)
    {
        k_ = (p - 1) >> BitShiftNum;
        a = modExp(r, k_, p);
        if (vec_idx < n)
        {
            w = 1;
            k = vec_idx & ((mid << 1) - 1);//  �����ת
            if (k < mid)
            {
                for (int l = 0; l < num_mini_batches; l++)
                {
                    factor1 = result[global_idx + mini_batch_offset * l];

                    uint128_t tmp;
                    mul64(twiddleFactorArray[(BitShiftNum - 1) * maxTwiddleCols + k], result[global_idx + mini_batch_offset * l + mid], tmp);
                    factor2 = (tmp % p).low;
                    output[global_idx + mini_batch_offset * l] = ((uint128_t(factor1) + factor2) % p).low;
                }
            }
            else
            {
                for (int l = 0; l < num_mini_batches; l++)
                {
                    factor1 = result[global_idx + mini_batch_offset * l - mid];
                    uint128_t tmp;
                    mul64(twiddleFactorArray[(BitShiftNum - 1) * maxTwiddleCols + k - mid], result[global_idx + mini_batch_offset * l], tmp);
                    factor2 = (tmp % p).low;
                    output[global_idx + mini_batch_offset * l] = ((uint128_t(factor1) + p - factor2) % p).low;
                }
            }
        }
        grid.sync();
        if (vec_idx < n)
            for (int l = 0; l < num_mini_batches; l++)
                result[global_idx + mini_batch_offset * l] = output[global_idx + mini_batch_offset * l];
        grid.sync();
    }
    result[global_idx] = result[global_idx]%n;
}