#include "hip/hip_runtime.h"
//Edit by Malone and Longson
//creat data:2023.1.1


#include<stdio.h>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#include <cmath>		
#include <cstdint> 		
#include <cstdlib>		
#include <ctime>		
#include <iostream> 		

#include "utils.cuh" 	
#include "uint128.h"
#include "parameters.h"

uint64_t ModularInv(uint64_t Data, uint64_t Mprime)
{
	uint128_t DataU = Mprime;
	uint128_t DataV = Data;
	uint128_t DataR = 0;
	uint128_t DataS = 1;
	uint128_t MprimeLocal = Mprime;
	uint32_t flagR = 0;

	while (DataV > 0)
	{
		if ((DataU.low & uint64_t(1)) == 0)
		{
			DataU = DataU >> 1;
			if ((DataR.low & uint64_t(1)) == 0)
				DataR = DataR >> 1;
			else
			{
				DataR = (DataR + MprimeLocal) >> 1;
			}
		}
		else if ((DataV.low & uint64_t(1)) == 0)
		{
			DataV = DataV >> 1;
			if ((DataS.low & uint64_t(1)) == 0)
				DataS = DataS >> 1;
			else
			{
				DataS = (DataS + MprimeLocal) >> 1;
			}

		}
		else
		{
			if (DataU > DataV)
			{

				DataU = DataU - DataV;

				if (DataR < DataS)
				{
					DataR = DataR + MprimeLocal - DataS;
				}
				else
					DataR = DataR - DataS;
			}
			else
			{
				DataV = DataV - DataU;

				if (DataS < DataR)
				{
					DataS = DataS + MprimeLocal - DataR;
				}
				else
					DataS = DataS - DataR;
			}
		}
	}

	if (DataU > 1) return 0;
	if (DataR > MprimeLocal) return (DataR - MprimeLocal).low;

	return DataR.low;

}

void preComputeTwiddleFactor(uint64_t* twiddleFactorArray, uint64_t n, uint64_t p, uint64_t r)
{
	uint64_t m = 1, a, k_;
	uint64_t w, z = 0;
	uint128_t  tmp;
	for (uint64_t mid = 1, BitShiftNum = 1; mid < n; mid = mid << 1, BitShiftNum++)
	{
		k_ = (p - 1) >> BitShiftNum;
		a = modExp(r, k_, p);
		for (uint64_t j = 0; j < n; j += (mid << 1)) {
			w = 1;
			for (uint64_t k = 0; k < mid; k++)
			{
				twiddleFactorArray[z] = w;
				z++;

				mul64(w, a, tmp);
				w = (tmp % p).low;

			}
		}
	}

}


void preComputeTwiddleFactor_step2nd(uint64_t* twiddleFactorArray, uint64_t Len_1D, uint64_t Len_2D, uint64_t p, uint64_t r, uint64_t wCoeff)
{

	uint64_t* twiddleFactorArrayPre = (uint64_t*)malloc(Len_1D * sizeof(uint64_t));


	for (int64_t ir = 0; ir < Len_1D; ir++)
	{
		twiddleFactorArrayPre[ir] = modExp(wCoeff, ir, p);
		twiddleFactorArray[ir] = 1;
		twiddleFactorArray[ir + Len_1D] = twiddleFactorArrayPre[ir];
	}

	uint128_t  tmp;
	for (int64_t ir = 2; ir < Len_2D; ir++)
	{
		for (int64_t ir2 = 0; ir2 < Len_1D; ir2++)
		{

			uint64_t Outtest;
			mul64modAdd(twiddleFactorArray[(ir - 1) * Len_1D + ir2], twiddleFactorArrayPre[ir2], 0, p, Outtest);
			twiddleFactorArray[ir * Len_1D + ir2] = Outtest;
		}

	}

	free(twiddleFactorArrayPre);
}

void DataReform(uint64_t* Data, uint64_t* DataOut, uint64_t Len_1D, uint64_t Len_2D)
{
	uint64_t* dataArray2 = (uint64_t*)calloc(Len_1D * Len_2D, sizeof(uint64_t));

	int64_t DataCnt = 0;
	uint64_t* DataSel = (uint64_t*)calloc(Len_2D, sizeof(uint64_t));

	for (uint64_t ir = 0; ir < Len_1D; ir++)
	{
		bit_reverse(Data + ir * Len_2D, DataSel, Len_2D);
		memcpy(dataArray2 + ir * Len_2D, DataSel, Len_2D * sizeof(uint64_t));
	}

	for (uint64_t ir2 = 0; ir2 < Len_2D; ir2++)
	{
		for (uint64_t ir = 0; ir < Len_1D; ir++)
		{
			DataOut[DataCnt] = dataArray2[ir * Len_2D + ir2];
			DataCnt++;
		}
	}

	free(DataSel);
	free(dataArray2);
}

bool compVec(uint64_t* vec1, uint64_t* vec2, uint64_t n, bool debug) {

	bool comp = true;
	for (uint64_t i = 0; i < n; i++) {

		if (vec1[i] != vec2[i]) {
			comp = false;

			if (debug) {
				std::cout << "(vec1[" << i << "] : " << vec1[i] << ")";
				std::cout << "!= (vec2[" << i << "] : " << vec2[i] << ")";
				std::cout << std::endl;
			}
			else {
				break;
			}
		}
	}

	return comp;
}

void bit_reverse(uint64_t* vec, uint64_t* vecOut, uint64_t n) {

	uint64_t num_bits = log2(n);

	uint64_t reverse_num;
	for (uint64_t i = 0; i < n; i++) {

		reverse_num = 0;
		for (uint64_t j = 0; j < num_bits; j++) {

			reverse_num = reverse_num << 1;
			if (i & (1 << j)) {
				reverse_num = reverse_num | 1;
			}
		}

		vecOut[reverse_num] = vec[i];

	}
}

void bit_reverseOfNumber(const uint64_t* Number, const uint64_t* nbit, uint64_t* reNumber)
{
	*reNumber = 0;
	uint64_t Temp = *Number;
	for (uint64_t ir = 0; ir < *nbit; ir++)
	{
		*reNumber = (*reNumber << 1) | ((Temp >> ir) & 1);
	}

	return;
}


__host__ __device__ uint64_t modExp(uint64_t base, uint64_t exp, uint64_t m) {

	uint64_t result = 1;
	uint128_t tmp;

	while (exp > 0) {

		if (exp % 2) {

			mul64(result, base, tmp);
			result = (tmp % m).low;

		}

		exp = exp >> 1;
		mul64(base, base, tmp);
		base = (tmp % m).low;
	}

	return result;
}


__host__ __device__ uint64_t modulo(int64_t base, int64_t m) {
	int64_t result = base % m;

	return (result >= 0) ? result : result + m;
}

void printVec(uint64_t* vec, uint64_t n) {

	std::cout << "[" << "\n";
	for (uint64_t i = 0; i < n; i++) {

		std::cout << vec[i] << "," << "\n";

	}
	std::cout << "]" << std::endl;
}

uint64_t* randVec(uint64_t n, uint64_t max) {

	uint64_t* vec;
	vec = (uint64_t*)malloc(n * sizeof(uint64_t));

	srand(time(0));
	for (uint64_t i = 0; i < n; i++) {

		vec[i] = rand() % (max + 1);
	}

	return vec;
}

void generateDate(uint64_t n, uint64_t* cpu_outdata)
{
	if (n >= 256)
	{
		uint64_t* cuda_outdata;
		hipMalloc(&cuda_outdata, n * sizeof(uint64_t));
		int tpb = THREDS_PER_BLOCK;
		int bpg = (n + 32) / THREDS_PER_BLOCK; 
		dim3 dimGrid(bpg, 1, 1);
		dim3 dimBlock(tpb, 1, 1);
		generate_data_kernal << <dimGrid, dimBlock >> > (cuda_outdata);
		hipMemcpyAsync(cpu_outdata, cuda_outdata, n * sizeof(uint64_t), hipMemcpyDeviceToHost);
		hipFree(cuda_outdata); 
	}
	else
	{
		for (int16_t ir = 0; ir < n; ir++)
		{
			cpu_outdata[ir] = ir + 1;
		}
	}

}

__global__ void generate_data_kernal(uint64_t* data)
{
	const int tid = blockDim.x * blockIdx.x + threadIdx.x;; 
	data[tid] = tid + 1;
}