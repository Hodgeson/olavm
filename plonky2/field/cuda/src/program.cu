
#include "program.cuh"

void cuda_vec_add(uint32_t *out, const uint32_t *a, const uint32_t *b, uint32_t size) {
    uint32_t *d_a, *d_b, *d_out;

    // malloc gpu memory and memcpy
    CHECK(hipMalloc((void **)&d_a, sizeof(uint32_t) * size));
    CHECK(hipMalloc((void **)&d_b, sizeof(uint32_t) * size));
    CHECK(hipMalloc((void **)&d_out, sizeof(uint32_t) * size));
    CHECK(hipMemcpy(d_a, a, sizeof(uint32_t) * size, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_b, b, sizeof(uint32_t) * size, hipMemcpyHostToDevice));

    // create event
    hipEvent_t start, end;
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&end));

    // warm up
    vec_add<<<128, 1024>>>(d_out, d_a, d_b, size);

    // run kernel
    float time_used;
    CHECK(hipEventRecord(start));
    vec_add<<<128, 1024>>>(d_out, d_a, d_b, size);
    CHECK(hipEventRecord(end));
    CHECK(hipDeviceSynchronize());
    CHECK(hipEventElapsedTime(&time_used, start, end));

    std::cout << "Time used: " << time_used << " ms." << std::endl;

    // get result from gpu
    CHECK(hipMemcpy(out, d_out, sizeof(uint32_t) * size, hipMemcpyDeviceToHost));
}